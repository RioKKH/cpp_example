#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CUDAKernels.hpp"
#include "Random123/philox.h"

typedef r123::Philox2x32 RNG_2x32;
typedef r123::Philox4x32 RNG_4x32;

__global__ void cudaKernelGenerateRandomNumber(unsigned int randomseed)
{
    uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    printf("%d\n", idx);
    RNG_4x32 rng_4x32;
    RNG_4x32::key_type key
        = {{static_cast<unsigned int>(threadIdx.x),
            static_cast<unsigned int>(blockIdx.x)}};
    RNG_4x32::ctr_type ctr = {{0, 0, randomseed, 0xbeeff00d}};
    RNG_4x32::ctr_type randomValues;

    ctr.incr();
    randomValues = rng_4x32(ctr, key);
    printf("%d,%d,%d,%d\n", randomValues[0], randomValues[1], randomValues[2], randomValues[3]);
    __syncthreads();
}


int main(int argc, char* argv[])
{
    unsigned int seed = 999;
    uint32_t blocknum = 16;
    uint32_t threadnum = 16;

    printf("Start\n");
    cudaKernelGenerateRandomNumber<<<blocknum, threadnum>>>(seed);
    printf("End\n");

    return 0;
}
