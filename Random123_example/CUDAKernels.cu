#include "hip/hip_runtime.h"
#include "Random123/philox.h"


using namespace r123;

/// Datatype for two 32b random values.
typedef r123::Philox2x32 RNG_2x32;

/// Datatype for four 32b random values.
typedef r123::Philox4x32 RNG_4x32;

/**
 * Generate two random values
 * @param [in] key     - Key for the generator.
 * @param [in] counter - Counter for the generator.
 * @return two random values
 */
__device__ RNG_2x32::ctr_type generateTwoRndValues(unsigned int key,
                                                   unsigned int counter);



/**
 * Device random number generator
 * 
 */
inline __device__ RNG_2x32::ctr_type generateTwoRndValues(unsigned int key,
                                                          unsigned int counter)
{
    RNG_2x32 rng;

    return rng({0, counter}, {key});
} // end of TwoRandomINTs

/**
 * Initialize Population before run.
 */
__global__ void cudaGenerateFirstPopulationKernel(unsigned int randomSeed)
{
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

    const int nGenes = 100;

    while (i < nGenes)
    {
        const RNG_2x32::ctr_type randomValues = generateTwoRndValues(i, randomSeed);


