#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Random123/philox.h"
#include "CUDAKernels.hpp"

using namespace r123;

/// Datatype for two 32b random values.
typedef r123::Philox2x32 RNG_2x32;

/// Datatype for four 32b random values.
typedef r123::Philox4x32 RNG_4x32;

// const float RANDMIN = -2147483648.0f;
// const float RANDMAX =  2147483647.0f;
const float RANDMAX = 4294967295.0f;

/** Generate two random values
 * @param [in] key     - Key for the generator.
 * @param [in] counter - Counter for the generator.
 * @return two random values
 */
__device__ RNG_2x32::ctr_type generateTwoRndValues(unsigned int key,
                                                   unsigned int counter);

/**
 * Device random number generator
 *                                   1
 */
inline __device__ RNG_2x32::ctr_type generateTwoRndValues(unsigned int key,
                                                          unsigned int counter)
{
    RNG_2x32 rng;

    return rng({0, counter}, {key});
} // end of TwoRandomINTs


/**
 * Initialize Population before run.
 */
__global__ void cudaGenerateRandomNumberKernel(float *rand1,
        float *rand2,
        unsigned int randomSeed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // const int nGenes = 100;

    // while (idx < nGenes)
    // {
    RNG_2x32::ctr_type randomValues;//  = generateTwoRndValues(idx, randomSeed);
                                    // const RNG_2x32::ctr_type randomValues = generateTwoRndValues(idx, randomSeed);
                                    // printf("%d,%d\n", randomValues.v[0], randomValues.v[1]);
                                    // printf("%d,%d\n", randomValues.v[0]+RANDMIN, randomValues.v[1]+RANDMIN);
    for (int i = 0; i < 5; ++i)
    {
        randomValues = generateTwoRndValues(idx+i, randomSeed);
        printf("%f,%f\n", float(randomValues.v[0]) / RANDMAX,
                float(randomValues.v[1]) / RANDMAX);
        // rand1[idx] = float(randomValues.v[0]) / RANDMAX;
        // rand2[idx] = float(randomValues.v[1]) / RANDMAX;
    }

    for (int i = 0; i < 5; ++i)
    {
        randomValues = generateTwoRndValues(idx, randomSeed+i);
        printf("%f,%f\n", float(randomValues.v[0]) / RANDMAX,
                          float(randomValues.v[1]) / RANDMAX);
    }
    // rand1[idx] = float(randomValues.v[0]);
    // rand2[idx] = float(randomValues.v[1]);
    // rand1[idx] = (float(randomValues.v[0]))/(RANDMAX - RANDMIN);
    // rand2[idx] = (float(randomValues.v[1]))/(RANDMAX - RANDMIN);
    // rand1[idx] = (float(randomValues.v[0]) + RANDMAX)/(RANDMAX - RANDMIN);
    // rand2[idx] = (float(randomValues.v[1]) + RANDMAX)/(RANDMAX - RANDMIN);
    // }
}

__global__ void cudaTest()
{
    printf("hello GPU\n");
}

