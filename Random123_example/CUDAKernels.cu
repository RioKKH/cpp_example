#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Random123/philox.h"
#include "CUDAKernels.hpp"

using namespace r123;

/// Datatype for two 32b random values.
typedef r123::Philox2x32 RNG_2x32;

/// Datatype for four 32b random values.
typedef r123::Philox4x32 RNG_4x32;

/**
 * Generate two random values
 * @param [in] key     - Key for the generator.
 * @param [in] counter - Counter for the generator.
 * @return two random values
 */
__device__ RNG_2x32::ctr_type generateTwoRndValues(unsigned int key,
                                                   unsigned int counter);

/**
 * Device random number generator
 * 
 */
inline __device__ RNG_2x32::ctr_type generateTwoRndValues(unsigned int key,
                                                          unsigned int counter)
{
    RNG_2x32 rng;

    return rng({0, counter}, {key});
} // end of TwoRandomINTs


/**
 * Initialize Population before run.
 */
__global__ void cudaGenerateRandomNumberKernel(float *rand1,
                                               float *rand2,
                                               unsigned int randomSeed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    const int nGenes = 100;

    // while (idx < nGenes)
    // {
    const RNG_2x32::ctr_type randomValues = generateTwoRndValues(idx, randomSeed);
    printf("%d,%d\n", randomValues.v[0], randomValues.v[1]);
    rand1[idx] = randomValues.v[0];
    rand2[idx] = randomValues.v[1];
    // }
}

__global__ void cudaTest()
{
    printf("hello GPU\n");
}

