#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "Random123/philox.h"

const int N = 96;

__global__ void generateRandomNumbers(uint32_t* output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("idx: %d\n", idx);
    r123::Philox4x32 rng;
    r123::Philox4x32::key_type key = {{idx, 0}};
    r123::Philox4x32::ctr_type ctr = {{0, 0}};

    auto result = rng(ctr, key);
    output[idx] = result.v[0];
    __syncthreads();
}

int main() {
    uint32_t* d_output;
    uint32_t* h_output[N];

    hipMalloc(&d_output, N * sizeof(uint32_t));
    generateRandomNumbers<<<N/32, 32>>>(d_output);
    hipMemcpy(h_output, d_output, N * sizeof(uint32_t), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        std::cout << h_output[i] << std::endl;
    }

    hipFree(d_output);
    return 0;
}
