#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CUDAKernels.hpp"

int main(int argc, char* argv[])
{
    int n; // number of device
    unsigned int N = 100;

    float *phost_rand1;
    float *phost_rand2;

    float *pdev_rand1;
    float *pdev_rand2;

    hipDeviceProp_t prop;
    int mMultiProcessorCount;
    int THREADS_NUM = 256;

    phost_rand1 = (float *)malloc(N * sizeof(float));
    phost_rand2 = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; ++i)
    {
        phost_rand1[i] = 0;
        phost_rand2[i] = 0;
    }

#ifdef _DEBUG
    for (int i = 0; i < N; ++i)
    {
        printf("%f, %f\n", phost_rand1[i], phost_rand2[i]);
    }
#endif // _DEBUG

    hipMalloc((void **)&pdev_rand1, N * sizeof(float));
    hipMalloc((void **)&pdev_rand2, N * sizeof(float));

    hipMemcpy(phost_rand1, pdev_rand1, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(phost_rand2, pdev_rand2, N * sizeof(float), hipMemcpyHostToDevice);

    hipGetDeviceCount(&n);
#ifdef _DEBUG
    for (int i = 0; i < n; ++i)
    {
        hipGetDeviceProperties(&prop, i);

        printf("device %d\n", i);
        printf("device name : %s\n", prop.name);
        printf("total global memory : %ld (MB)\n", prop.totalGlobalMem/1024/1024);
        printf("shared memory : %ld (KB)\n", prop.sharedMemPerBlock/1024);
        printf("register / block : %d\n", prop.regsPerBlock);
        printf("warp size : %d\n", prop.warpSize);
        printf("max pitch : %ld (B)\n", prop.memPitch);
        printf("max thread / block : %d\n", prop.maxThreadsPerBlock);
        printf("max size of each dim. of block : (%d, %d, %d)\n", 
                prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("max size of each dim. of grid : (%d, %d, %d)\n", 
                prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("clock rate : %d (MHz)\n", prop.clockRate);
        printf("total constant memory : %ld (KB)\n", prop.totalConstMem);
        printf("comute capatility  : %d.%d\n", prop.major, prop.minor);
        printf("alignment requirement for texture : %ld\n", prop.textureAlignment);
        printf("device overlap : %s\n", (prop.deviceOverlap ? "OK" : "NOT"));
        printf("num. of multiprocessors : %d\n", prop.multiProcessorCount);
        printf("kernel execution timeout : %s\n", (prop.kernelExecTimeoutEnabled ? "ON" : "OFF"));
        printf("integrate : %s\n", (prop.integrated ? "ON" : "OFF"));
        printf("host memory mapping : %s\n", (prop.canMapHostMemory ? "ON" : "OFF"));
        printf("compute mode : \n");
        if (prop.computeMode == hipComputeModeDefault) printf("default mode (multiple threads can be used\n");
        else if (prop.computeMode == hipComputeModeExclusive) printf("exclusive mode (only one thread can be used)\n");
        else if (prop.computeMode == hipComputeModeProhibited) printf("prohibited mode (no threads can be used\n)");
    }
#endif
    mMultiProcessorCount = prop.multiProcessorCount;

    // cudaGenerateRandomNumberKernelTest<<<mMultiProcessorCount * 2, THREADS_NUM>>>();
    printf("pre\n");
    // cudaTest<<<1, 1>>>();
    cudaGenerateRandomNumberKernel<<<10, 10>>>(pdev_rand1, pdev_rand2, N);
    hipDeviceSynchronize();
    printf("post\n");
    // cudaGenerateRandomNumberKernel<<<mMultiProcessorCount * 2, THREADS_NUM>>>(pdev_rand1, pdev_rand2, N);

    hipFree(pdev_rand1);
    hipFree(pdev_rand2);

    return 0;
}
