#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

const int N = 1024;

__global__ void generateRandomNumbers(hiprandStatePhilox4_32_10_t* states, float* output) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(tid, 0, 0, &states[tid]);
    output[tid] = hiprand_uniform(&states[tid]);
}

int main() {
    hiprandStatePhilox4_32_10_t* d_states;
    float h_output[N];

    hipMalloc(&d_states, N * sizeof(hiprandStatePhilox4_32_10_t));
    // cudaMalloc(&h_output, N * sizeof(float));
    generateRandomNumbers<<<N/256, 256>>>(d_states, h_output);
    hipMemcpy(h_output, d_states, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        std::cout << h_output[i] << std::endl;
    }

    hipFree(d_states);
    hipFree(h_output);
    return 0;
}
